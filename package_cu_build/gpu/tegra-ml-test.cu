#include <stdio.h>
#include <string.h>
#include "gpu_info_tegra.h"

int main() {

    size_t freeMem, totalMem;

    // cudaDeviceCount -> nvmlDeviceGetCount_v2
    int deviceCount; // 

    // hipGetDeviceProperties -> nvmlDeviceGetCudaComputeCapability
    int major = 0;
    int minor = 0;

    // Execute test functions
    // hipSetDevice
    hipError_t cudaInitStatus = hipSetDevice(0);

    // hipDeviceReset
    hipError_t cudaResetStatus = hipDeviceReset();

    // nvmlDeviceGetHandleByIndex -- Directly store the index as the "handle". Not directly used.
    // handle_device = (nvmlDevice_t)index;

    // hipMemGetInfo
    hipError_t cudaMemInfoStatus = hipMemGetInfo(&freeMem, &totalMem);
    hipError_t cudaDeviceCountStatus = hipGetDeviceCount(&deviceCount);

    // hipGetDeviceProperties -> nvmlDevicesComputeCapability
    hipDeviceProp_t deviceProp;
    int device_handle = 0;
    hipError_t cudaDevicePropertiesStatus = hipGetDeviceProperties(&deviceProp, device_handle);
    
    major = deviceProp.major;
    minor = deviceProp.minor;

    unsigned long total_mb = totalMem / (1024 * 1024);
    unsigned long free_mb = freeMem / (1024 * 1024);
    unsigned long used_mb = (totalMem - freeMem) / (1024 * 1024);
    unsigned long usedMem = (totalMem - freeMem);

    printf("Device Number: %d || nvmlInit_v2() and nvmlShutDown() good.\n", device_handle);
    printf("  Memory info:\n");
    printf("    Total: %lu (%lu MB)\n", totalMem, total_mb);
    printf("    Used:  %lu (%lu MB)\n", usedMem, used_mb);
    printf("    Free:  %lu (%lu MB)\n", freeMem, free_mb);
    printf("\n");
    printf("  Device Count: %d\n", deviceCount);
    printf("  CUDA Compute Capability: %d.%d\n", major, minor);

}