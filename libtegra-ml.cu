#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu_info_tegra.h"

extern "C" {
    nvmlReturn_t nvmlInit_v2(void *) {
        hipError_t cudaStatus = hipSetDevice(0); // Sets the first device as active
        if (cudaStatus != hipSuccess) {
            return NVML_UNKNOWN_ERROR; // Map CUDA error to NVML error
        }
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlShutdown(void *) {
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            return NVML_UNKNOWN_ERROR;
        }
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlDeviceGetHandleByIndex(unsigned int index, nvmlDevice_t *device) {
        // In CUDA Runtime API, devices are typically referred to by their index
        *device = (nvmlDevice_t)index; // Directly store the index as the "handle"
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlDeviceGetMemoryInfo(nvmlDevice_t device, nvmlMemory_t *memory) {
        size_t freeMem, totalMem;
        hipError_t cudaStatus = hipMemGetInfo(&freeMem, &totalMem);
        if (cudaStatus != hipSuccess) {
            return NVML_UNKNOWN_ERROR;
        }
        memory->total = totalMem;
        memory->free = freeMem;
        memory->used = totalMem - freeMem;
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlDeviceGetCount_v2(unsigned int *deviceCount) {
        int count;
        hipError_t cudaStatus = hipGetDeviceCount(&count);
        if (cudaStatus != hipSuccess) {
            return NVML_UNKNOWN_ERROR;
        }
        *deviceCount = (unsigned int)count;
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlDeviceGetCudaComputeCapability(nvmlDevice_t device, int *major, int *minor) {
        hipDeviceProp_t deviceProp;
        hipError_t cudaStatus = hipGetDeviceProperties(&deviceProp, device);
        if (cudaStatus != hipSuccess) {
            return NVML_UNKNOWN_ERROR;
        }
        *major = deviceProp.major;
        *minor = deviceProp.minor;
        return NVML_SUCCESS;
    }

}