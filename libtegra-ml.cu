#include <stdio.h>

nvmlReturn_t tegraInit_v2(void *) {
    hipError_t cudaStatus = hipSetDevice(0); // Sets the first device as active
    if (cudaStatus != hipSuccess) {
        return NVML_UNKNOWN_ERROR; // Map CUDA error to NVML error
    }
    return NVML_SUCCESS;
}

nvmlReturn_t tegraShutdown(void *) {
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        return NVML_UNKNOWN_ERROR;
    }
    return NVML_SUCCESS;
}

nvmlReturn_t tegraDeviceGetHandleByIndex(unsigned int index, nvmlDevice_t *device) {
    // In CUDA Runtime API, devices are typically referred to by their index
    *device = (nvmlDevice_t)index; // Directly store the index as the "handle"
    return NVML_SUCCESS;
}

nvmlReturn_t tegraDeviceGetMemoryInfo(nvmlDevice_t device, nvmlMemory_t *memory) {
    size_t freeMem, totalMem;
    hipError_t cudaStatus = hipMemGetInfo(&freeMem, &totalMem);
    if (cudaStatus != hipSuccess) {
        return NVML_UNKNOWN_ERROR;
    }
    memory->total = totalMem;
    memory->free = freeMem;
    memory->used = totalMem - freeMem;
    return NVML_SUCCESS;
}

nvmlReturn_t tegraDeviceGetCount_v2(unsigned int *deviceCount) {
    int count;
    hipError_t cudaStatus = hipGetDeviceCount(&count);
    if (cudaStatus != hipSuccess) {
        return NVML_UNKNOWN_ERROR;
    }
    *deviceCount = (unsigned int)count;
    return NVML_SUCCESS;
}

nvmlReturn_t tegraDeviceGetCudaComputeCapability(nvmlDevice_t device, int *major, int *minor) {
    hipDeviceProp_t deviceProp;
    int device_handle = 0;
    hipError_t cudaStatus = hipGetDeviceProperties(&deviceProp, device_handle);
    if (cudaStatus != hipSuccess) {
        return NVML_UNKNOWN_ERROR;
    }
    *major = deviceProp.major;
    *minor = deviceProp.minor;
    return NVML_SUCCESS;
}
