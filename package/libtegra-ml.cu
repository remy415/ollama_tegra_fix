#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu_info_tegra.h"

extern "C" {
    nvmlReturn_t nvmlInit_v2(void *) {
        hipError_t cudaStatus = hipSetDevice(0); // Sets the first device as active
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed: %s\n", hipGetErrorString(cudaStatus));
            return NVML_UNKNOWN_ERROR; // Map CUDA error to NVML error
        }
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlShutdown(void *) {
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed: %s\n", hipGetErrorString(cudaStatus));
            return NVML_UNKNOWN_ERROR;
        }
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlDeviceGetHandleByIndex(unsigned int index, nvmlDevice_t *device) {
        // In CUDA Runtime API, devices are typically referred to by their index
        *device = (nvmlDevice_t)index; // Directly store the index as the "handle"
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlDeviceGetMemoryInfo(nvmlDevice_t device, nvmlMemory_t *memory) {
        size_t freeMem, totalMem;
        hipError_t cudaStatus = hipMemGetInfo(&freeMem, &totalMem);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemGetInfo failed: %s\n", hipGetErrorString(cudaStatus));
            return NVML_UNKNOWN_ERROR;
        }
        memory->total = totalMem;
        memory->free = freeMem;
        memory->used = totalMem - freeMem;
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlDeviceGetCount_v2(unsigned int *deviceCount) {
        int count;
        hipError_t cudaStatus = hipGetDeviceCount(&count);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(cudaStatus));
            return NVML_UNKNOWN_ERROR;
        }
        *deviceCount = (unsigned int)count;
        return NVML_SUCCESS;
    }

    nvmlReturn_t nvmlDeviceGetCudaComputeCapability(nvmlDevice_t device, int *major, int *minor) {
        hipDeviceProp_t deviceProp;
        int device_handle = 0;
        hipError_t cudaStatus = hipGetDeviceProperties(&deviceProp, device_handle);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGetDeviceProperties failed: %s\n", hipGetErrorString(cudaStatus));
            return NVML_UNKNOWN_ERROR;
        }
        *major = deviceProp.major;
        *minor = deviceProp.minor;
        return NVML_SUCCESS;
    }

}